#include "hip/hip_runtime.h"
/*
 * Parallel path tracer with CUDA.
 */

#include <iostream>
#include <float.h>

// CUDA libraries
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "surface_list.h"
#include "camera.h"
#include "material.h"

///////////////////////////////////////////////////////////////////////////////////

__device__ vec3 color(const ray& r, hittable **world, hiprandState *local_rand_state, int depth) {
  hit_record rec;

  if ((*world)->hit(r, 0.001f, FLT_MAX, rec)) {
    ray scattered;
    vec3 attenuation;

    if (depth < 50 && rec.mat_ptr->scatter(r, rec, attenuation, scattered, local_rand_state)){
      return attenuation*color(scattered, world, local_rand_state, depth+1);
    }else {
      return vec3(0, 0, 0);
    }

  }else {
      vec3 unit_direction = unit_vector(r.direction());
      float t = 0.5f*(unit_direction.y() + 1.0f);

      return (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
  }
}

/*
 * CUDA kernel: initialize rand_state, separated from 
 * actual rendering for performance measurement
 */
__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= max_x) || (j >= max_y)) return;
  int pixel_index = j*max_x + i;

  // Each thread gets same seed, a different sequence number, no offset
  hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}


/*
 * CUDA kernel function
 */
__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam,
		       hittable **world, hiprandState *rand_state) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  // Only run on pixels within the image
  if((i >= max_x) || (j >= max_y)) return;

  int pixel_index = j*max_x + i;
  vec3 col(0, 0, 0);
  
  // Local copy of random state
  hiprandState local_rand_state = rand_state[pixel_index];
  
  for (int s = 0; s < ns; s++){
    float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
    float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);

    ray r = (*cam)->get_ray(u,v);
    col += color(r, world, &local_rand_state, 0);
  }

  rand_state[pixel_index] = local_rand_state;
  col /= float(ns);
  col[0] = sqrt(col[0]);
  col[1] = sqrt(col[1]);
  col[2] = sqrt(col[2]);
  fb[pixel_index] = col;
}

/*
 * CUDA kernel: construct scene's objects
 */
__global__ void create_world(hittable **d_list, hittable **d_world, camera **d_camera) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    //    *(d_list)   = new sphere(vec3(0,0,-1), 0.5, new matte(vec3(0.4, 0.2, 0.1)));
    *(d_list) = new sphere(vec3(2,0,-1), 0.5, new metal(vec3(0.7, 0.6, 0.5), 0.0));
    *(d_list+1) = new sphere(vec3(0,-100.5,-1), 100, new matte(vec3(0.5, 0.5, 0.5)));
    *d_world    = new surface_list(d_list, 2);
    *d_camera   = new camera();
  }
}

/*
 * CUDA kernel: deallocate scene's objects
 */
__global__ void free_world(hittable **d_list, hittable **d_world, camera **d_camera) {
  delete *(d_list);
  delete *(d_list+1);
  delete *d_world;
  delete *d_camera;
}

/********************* MAIN ******************************************************/

int main() {

  int nx = 600; // image width
  int ny = 300;  // image height
  int tx = 8;    // block width
  int ty = 8;    // block height
  int ns = 50;  // number of samples
  
  int num_pixels = nx*ny;
  size_t fb_size = num_pixels*sizeof(vec3);
  
  std::cerr << "--------------------------------------------------------------\n\n";
  std::cerr << "Rendering a " << nx << "x" << ny << " image with "
	    << ns << " samples per pixel. \n";
  std::cerr << tx << "x" << ty << " blocks.\n";

  // Allocate unified memory for frame buffer
  vec3 *fb;
  checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

  // Allocate random state
  hiprandState *d_rand_state;
  checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
  
  // Allocate world of objects and the camera
  hittable **d_list;
  checkCudaErrors(hipMalloc((void **)&d_list, 2*sizeof(hittable *)));
  hittable **d_world;
  checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));
  camera **d_camera;
  checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
  create_world<<<1,1>>>(d_list, d_world, d_camera);

  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  
  // For timing execution of kernel code
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  // Dimensions of grids and blocks
  dim3 dimGrid(nx/tx+1,ny/ty+1);
  dim3 dimBlock(tx,ty);

  render_init<<<dimGrid, dimBlock>>>(nx, ny, d_rand_state);
  
  checkCudaErrors(hipEventRecord(start));
  
  // Kernel invocation
  render<<<dimGrid, dimBlock>>>(fb, nx, ny, ns, d_camera, d_world, d_rand_state);

  checkCudaErrors(hipGetLastError());

  // Wait for GPU to finish
  checkCudaErrors(hipDeviceSynchronize());
  
  // Print out performance metrics
  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));
  float ms = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&ms, start, stop));
  std::cerr << "PERFORMANCE: " << ms << " ms\n\n";
  std::cerr << "--------------------------------------------------------------\n";

  // Output frame buffer as image
  std::cout << "P3\n" << nx << " " << ny << "\n255\n";
  for (int j = ny-1; j >= 0; j--) {
    for (int i = 0; i < nx; i++) {

      size_t pixel_index = j*nx + i;

      int ir = int(255.99*fb[pixel_index].r());
      int ig = int(255.99*fb[pixel_index].g());
      int ib = int(255.99*fb[pixel_index].b());

      std::cout << ir << " " << ig << " " << ib << "\n";

    }
  }

  // CUDA clean-up
  checkCudaErrors(hipDeviceSynchronize());
  free_world<<<1, 1>>>(d_list,d_world, d_camera);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipFree(d_list));
  checkCudaErrors(hipFree(d_world));
  checkCudaErrors(hipFree(d_camera));
  checkCudaErrors(hipFree(d_rand_state));
  checkCudaErrors(hipFree(fb));
  
}
